
#include <hip/hip_runtime.h>
#ifdef __NVCC__

#include "Vector/vector_dist.hpp"

template<typename vector_type, typename NN_type>
__global__ void probe_neighbors(vector_type vd, NN_type NN)
{
    auto a = GET_PARTICLE(vd);
    if (a == 10) {
        auto itn = NN.getNNIteratorBoxSym(a, NN.getCell(vd.getPos(a)));
        //Point<2,double> xp({1.0, 1.0});
        //auto itn = NN.getNNIteratorBox(NN.getCell(xp));
        while (itn.isNext()) {
            auto b = itn.get();
            if (a != b)	{
                vd.template getProp<0>(b) = 1.0;
            }
            ++itn;
        }
    }
}

int main(int argc, char* argv[])
{
	openfpm_init(&argc,&argv);

    size_t sz[2] = {10, 10};
    size_t bc[2] = {NON_PERIODIC, NON_PERIODIC};
    Box<2, double> domain({0, 0}, {2.0, 2.0});
    Ghost<2, double> g(0.1);
    vector_dist_gpu<2, double, aggregate<double>> vd(0, domain, bc, g);

	auto grid_it = vd.getGridIterator(sz);
    while (grid_it.isNext()) {
        vd.add();

        auto key = grid_it.get();
        double x = key.get(0) * grid_it.getSpacing(0);
        double y = key.get(1) * grid_it.getSpacing(1);
        double z = key.get(2) * grid_it.getSpacing(2);

        vd.getLastPos()[0] = x;
        vd.getLastPos()[1] = y;
        vd.getLastPos()[2] = z;

        vd.template getLastProp<0>() = -1.0;
        ++grid_it;
    }

    auto NN_sym = vd.getCellListGPU(0.4, CL_SYMMETRIC);
    auto NN_nonsym = vd.getCellListGPU(0.4, CL_NON_SYMMETRIC);
    vd.updateCellListGPU(NN_sym);
    vd.updateCellListGPU(NN_nonsym);
    NN_sym.debug_deviceToHost();
    NN_nonsym.debug_deviceToHost();
    std::cout << "Number of cells :" << NN_sym.getNCells() << " " << NN_nonsym.getNCells() << std::endl;
    
    std::cout << "Elements distribution across NN_nonsym cells:" << std::endl;
    for (int i = 0; i < NN_nonsym.getNCells(); i++) {
        if (NN_nonsym.getNelements(i) > 0) {
            std::cout << i << ": " << NN_nonsym.getNelements(i) << std::endl;
        }
    }

    std::cout << "Elements distribution across NN_sym cells:" << std::endl;
    for (int i = 0; i < NN_sym.getNCells(); i++) {
        if (NN_sym.getNelements(i) > 0) {
            std::cout << i << ": " << NN_sym.getNelements(i) << std::endl;
        }
    }
    
    auto vd_it_gpu = vd.getDomainIteratorGPU();
    CUDA_LAUNCH(probe_neighbors, vd_it_gpu, vd.toKernel(), NN_sym.toKernel());
    vd.template deviceToHostProp<0>();
    
    auto vd_it = vd.getDomainIterator();
    while (vd_it.isNext()) {
        auto key = vd_it.get();
        std::cout << vd.getPos(key)[0] << " " << vd.getPos(key)[1] << " " << vd.getProp<0>(key) << std::endl;
        ++vd_it;
    }

	openfpm_finalize();
}
 
#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif