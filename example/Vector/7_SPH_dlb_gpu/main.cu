
#include <hip/hip_runtime.h>
/*! \page Vector_7_sph_dlb_gpu Vector 7 SPH Dam break simulation with Dynamic load balacing on Multi-GPU
 *
 *
 * [TOC]
 *
 *
 * # SPH with Dynamic load Balancing on GPU # {#SPH_dlb_gpu}
 *
 *
 * This example show the classical SPH Dam break simulation with load balancing and dynamic load balancing. The main difference with
 * \ref SPH_dlb is that here we use GPUs and 1.2 Millions particles.
 *
 * \htmlonly
 * <a href="#" onclick="hide_show('vector-video-3')" >Simulation video 1</a><br>
 * <div style="display:none" id="vector-video-3">
 * <video id="vid3" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu1.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-4')" >Simulation video 2</a><br>
 * <div style="display:none" id="vector-video-4">
 * <video id="vid4" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu2.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-15')" >Simulation video 3</a><br>
 * <div style="display:none" id="vector-video-15">
 * <video id="vid15" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu3.mp4" type="video/mp4"></video>
 * </div>
 * \endhtmlonly
 *
 * This example use all the features explained in example \ref e3_md_gpu. Additionally this example show how to remove particles
 * on GPU using a bulk remove function on GPU
 *
 * ## Bulk remove
 *
 * On SPH we have the necessity to remove particles that go out of bound. OpenFPM provide the function \b remove_marked \b .
 *
 * \snippet Vector/7_SPH_dlb_gpu/main.cu remove_marked_part
 *
 * where vd is the vector_dist_gpu red is the property that mark which particle must be removed. We mark the particle to be removed in the function kernel
 * We check if the particle go out of the region of interest or their density go critically far from the rest density
 *
 * \snippet Vector/7_SPH_dlb_gpu/main.cu mark_to_remove_kernel
 *
 * ## Macro CUDA_LAUNCH
 *
 * When we want to launch a kernel "my_kernel" on CUDA we in general use the Nvidia CUDA syntax
 *
 * my_kernel<<<wthr,thr>>>(arguments ... )
 *
 * Where wthr is the number of workgroups and thr is the number of threads in a workgroup and arguments... are the arguments to pass to the kernel. 
 * Equivalently we can launch a kernel with the macro CUDA_LAUNCH_DIM3(my_kernel,wthr,thr,arguments...) or CUDA_LAUNCH(my_kernel,ite,arguments) where
 * ite has been taken using getDomainIteratorGPU. There are several advantage on using CUDA_LAUNCH. The first advantage in using the macro is enabling SE_CLASS1
 * all kernel launch become synchronous and an error check is performed before continue to the next kernel making debugging easier. Another feature is the possibility
 * to run CUDA code on CPU without a GPU. compiling with "CUDA_ON_CPU=1 make" (Note openfpm must be compiled with GPU support (-g)  or with CUDA_ON_CPU support 
 * (-c "... --enable_cuda_on_cpu"). You can compile this example on CPU. You do not have to change a single line of code for this example. (Check the video to see this 
 * feature in action). All the openfpm GPU example and CUDA example can run on CPU if they use CUDA_LAUNCH as macro. We are planning to support
 * AMD GPUs as well using this system.
 *
 * \include Vector/7_SPH_dlb_gpu_opt/main.cu
 *
 */

#ifdef __NVCC__

#include "Vector/vector_dist.hpp"
#include <math.h>
#include "Draw/DrawParticles.hpp"

typedef float real_number;

// A constant to indicate boundary particles
#define BOUNDARY 0

// A constant to indicate fluid particles
#define FLUID 1

// initial spacing between particles dp in the formulas
const real_number dp = 0.0085;
// Maximum height of the fluid water
// is going to be calculated and filled later on
real_number h_swl = 0.0;

// c_s in the formulas (constant used to calculate the sound speed)
const real_number coeff_sound = 20.0;

// gamma in the formulas
const real_number gamma_ = 7.0;

// sqrt(3.0*dp*dp) support of the kernel
const real_number H = 0.0147224318643;

// Eta in the formulas
const real_number Eta2 = 0.01 * H*H;

// alpha in the formula
const real_number visco = 0.1;

// cbar in the formula (calculated later)
real_number cbar = 0.0;

// Mass of the fluid particles
const real_number MassFluid = 0.000614125;

// Mass of the boundary particles
const real_number MassBound = 0.000614125;

// End simulation time
#ifdef TEST_RUN
const real_number t_end = 0.001;
#else
const real_number t_end = 1.5;
#endif

// Gravity acceleration
const real_number gravity = 9.81;

// Reference densitu 1000Kg/m^3
const real_number rho_zero = 1000.0;

// Filled later require h_swl, it is b in the formulas
real_number B = 0.0;

// Constant used to define time integration
const real_number CFLnumber = 0.2;

// Minimum T
const real_number DtMin = 0.00001;

// Minimum Rho allowed
const real_number RhoMin = 700.0;

// Maximum Rho allowed
const real_number RhoMax = 1300.0;

// Filled in initialization
real_number max_fluid_height = 0.0;

// Properties

// FLUID or BOUNDARY
const size_t type = 0;

// Density
const int rho = 1;

// Density at step n-1
const int rho_prev = 2;

// Pressure
const int Pressure = 3;

// Delta rho calculated in the force calculation
const int drho = 4;

// calculated force
const int force = 5;

// velocity
const int velocity = 6;

// velocity at previous step
const int velocity_prev = 7;

const int red = 8;

const int red2 = 9;

// Type of the vector containing particles
typedef vector_dist_gpu<3,real_number,aggregate<size_t,real_number,  real_number,    real_number,     real_number,     real_number[3], real_number[3], real_number[3], real_number, real_number>> particles;
//                                              |          |             |               |                |                |               |               |               |            |
//                                              |          |             |               |                |                |               |               |               |            |
//                                             type      density       density        Pressure          delta            force          velocity        velocity        reduction     another
//                                                                     at n-1                           density                                         at n - 1        buffer        reduction buffer


struct ModelCustom
{
	template<typename Decomposition, typename vector> inline void addComputation(Decomposition & dec,
			                                                                     vector & vd,
																				 size_t v,
																				 size_t p)
	{
		if (vd.template getProp<type>(p) == FLUID)
			dec.addComputationCost(v,4);
		else
			dec.addComputationCost(v,3);
	}

	template<typename Decomposition> inline void applyModel(Decomposition & dec, size_t v)
	{
		dec.setSubSubDomainComputationCost(v, dec.getSubSubDomainComputationCost(v) * dec.getSubSubDomainComputationCost(v));
	}

	real_number distributionTol()
	{
		return 1.01;
	}
};

template<typename vd_type>
__global__ void EqState_gpu(vd_type vd, real_number B)
{
	auto a = GET_PARTICLE(vd);

	real_number rho_a = vd.template getProp<rho>(a);
	real_number rho_frac = rho_a / rho_zero;

	vd.template getProp<Pressure>(a) = B*( rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac - 1.0);
}

inline void EqState(particles & vd)
{
	auto it = vd.getDomainIteratorGPU();

	// You can use standard CUDA kernel launch or the macro CUDA_LAUNCH

	//EqState_gpuning<<<it.wthr,it.thr>>>(vd.toKernel(),B);
	CUDA_LAUNCH(EqState_gpu,it,vd.toKernel(),B);
}


const real_number a2 = 1.0/M_PI/H/H/H;

inline __device__ __host__ real_number Wab(real_number r)
{
	r /= H;

	if (r < 1.0)
		return (1.0 - 3.0/2.0*r*r + 3.0/4.0*r*r*r)*a2;
	else if (r < 2.0)
		return (1.0/4.0*(2.0 - r*r)*(2.0 - r*r)*(2.0 - r*r))*a2;
	else
		return 0.0;
}


const real_number c1 = -3.0/M_PI/H/H/H/H;
const real_number d1 = 9.0/4.0/M_PI/H/H/H/H;
const real_number c2 = -3.0/4.0/M_PI/H/H/H/H;
const real_number a2_4 = 0.25*a2;
// Filled later
real_number W_dap = 0.0;

inline __device__ __host__ void DWab(Point<3,real_number> & dx, Point<3,real_number> & DW, real_number r, bool print)
{
	const real_number qq=r/H;

    real_number qq2 = qq * qq;
    real_number fac1 = (c1*qq + d1*qq2)/r;
    real_number b1 = (qq < 1.0)?1.0f:0.0f;

    real_number wqq = (2.0 - qq);
    real_number fac2 = c2 * wqq * wqq / r;
    real_number b2 = (qq >= 1.0 && qq < 2.0)?1.0f:0.0f;

    real_number factor = (b1*fac1 + b2*fac2);

    DW.get(0) = factor * dx.get(0);
    DW.get(1) = factor * dx.get(1);
    DW.get(2) = factor * dx.get(2);
}

// Tensile correction
inline __device__ __host__  real_number Tensile(real_number r, real_number rhoa, real_number rhob, real_number prs1, real_number prs2, real_number W_dap)
{
	const real_number qq=r/H;
	//-Cubic Spline kernel
	real_number wab;
	if(r>H)
	{
		real_number wqq1=2.0f-qq;
		real_number wqq2=wqq1*wqq1;

		wab=a2_4*(wqq2*wqq1);
	}
	else
	{
	    real_number wqq2=qq*qq;
	    real_number wqq3=wqq2*qq;

	    wab=a2*(1.0f-1.5f*wqq2+0.75f*wqq3);
	}

	//-Tensile correction.
	real_number fab=wab*W_dap;
	fab*=fab; fab*=fab; //fab=fab^4
	const real_number tensilp1=(prs1/(rhoa*rhoa))*(prs1>0? 0.01: -0.2);
	const real_number tensilp2=(prs2/(rhob*rhob))*(prs2>0? 0.01: -0.2);

	return (fab*(tensilp1+tensilp2));
}


inline __device__ __host__ real_number Pi(const Point<3,real_number> & dr, real_number rr2, Point<3,real_number> & dv, real_number rhoa, real_number rhob, real_number massb, real_number cbar, real_number & visc)
{
	const real_number dot = dr.get(0)*dv.get(0) + dr.get(1)*dv.get(1) + dr.get(2)*dv.get(2);
	const real_number dot_rr2 = dot/(rr2+Eta2);
	visc=(dot_rr2 < visc)?visc:dot_rr2;

	if(dot < 0)
	{
		const float amubar=H*dot_rr2;
		const float robar=(rhoa+rhob)*0.5f;
		const float pi_visc=(-visco*cbar*amubar/robar);

		return pi_visc;
    }
	else
		return 0.0;
}

template<typename particles_type, typename NN_type>
__global__ void calc_forces_gpu(particles_type vd, NN_type NN, real_number W_dap, real_number cbar)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	real_number max_visc = 0.0;

	// Get the position xp of the particle
	Point<3,real_number> xa = vd.getPos(a);

	// Take the mass of the particle dependently if it is FLUID or BOUNDARY
	real_number massa = (vd.template getProp<type>(a) == FLUID)?MassFluid:MassBound;

	// Get the density of the of the particle a
	real_number rhoa = vd.template getProp<rho>(a);

	// Get the pressure of the particle a
	real_number Pa = vd.template getProp<Pressure>(a);

	// Get the Velocity of the particle a
	Point<3,real_number> va = vd.template getProp<velocity>(a);

	// Reset the force counter (- gravity on zeta direction)
	vd.template getProp<force>(a)[0] = 0.0;
	vd.template getProp<force>(a)[1] = 0.0;
	vd.template getProp<force>(a)[2] = -gravity;
	vd.template getProp<drho>(a) = 0.0;

	// We threat FLUID particle differently from BOUNDARY PARTICLES ...
	if (vd.template getProp<type>(a) != FLUID)
	{

		// If it is a boundary particle calculate the delta rho based on equation 2
		// This require to run across the neighborhoods particles of a
		auto Np = NN.getNNIteratorBoxSym(a, NN.getCell(vd.getPos(a)));

		// For each neighborhood particle
		while (Np.isNext() == true)
		{
			// ... q
			auto b = Np.get();

			// Get the position xp of the particle
			Point<3,real_number> xb = vd.getPos(b);

			// if (p == q) skip this particle
			if (a == b)	{++Np; continue;};

			// get the mass of the particle
			real_number massb = (vd.template getProp<type>(b) == FLUID)?MassFluid:MassBound;

			// Get the velocity of the particle b
			Point<3,real_number> vb = vd.template getProp<velocity>(b);

			// Get the pressure and density of particle b
			real_number Pb = vd.template getProp<Pressure>(b);
			real_number rhob = vd.template getProp<rho>(b);

			// Get the distance between p and q
			Point<3,real_number> dr = xa - xb;
			Point<3,real_number> minus_dr = xb - xa;
			// take the norm of this vector
			real_number r2 = norm2(dr);

			// If the particles interact ...
			if (r2 < 4.0*H*H)
			{
				// ... calculate delta rho
				real_number r = sqrt(r2);

				Point<3,real_number> dv = va - vb;
				Point<3,real_number> minus_dv = vb - va;

				Point<3,real_number> DW;
				DWab(dr,DW,r,false);

				Point<3,real_number> minus_DW;
				DWab(minus_dr,minus_DW,r,false);

				const real_number dot = dr.get(0)*dv.get(0) + dr.get(1)*dv.get(1) + dr.get(2)*dv.get(2);
				const real_number dot_rr2 = dot/(r2+Eta2);
				max_visc = (dot_rr2 < max_visc)?max_visc:dot_rr2;

				vd.template getProp<drho>(a) += massb*(dv.get(0)*DW.get(0)+dv.get(1)*DW.get(1)+dv.get(2)*DW.get(2));
				atomicAdd(&vd.template getProp<drho>(b), massa*(minus_dv.get(0)*minus_DW.get(0)+minus_dv.get(1)*minus_DW.get(1)+minus_dv.get(2)*minus_DW.get(2)));
			}

			++Np;
		}

		vd.template getProp<red>(a) = max_visc;
	}
	else
	{
		// If it is a fluid particle calculate based on equation 1 and 2

		// Get an iterator over the neighborhood particles of p
		auto Np = NN.getNNIteratorBoxSym(a, NN.getCell(vd.getPos(a)));

		// For each neighborhood particle
		while (Np.isNext() == true)
		{
			// ... q
			auto b = Np.get();

			// Get the position xp of the particle
			Point<3,real_number> xb = vd.getPos(b);

			// if (p == q) skip this particle
			if (a == b)	{++Np; continue;};

			real_number massb = (vd.template getProp<type>(b) == FLUID)?MassFluid:MassBound;
			Point<3,real_number> vb = vd.template getProp<velocity>(b);
			real_number Pb = vd.template getProp<Pressure>(b);
			real_number rhob = vd.template getProp<rho>(b);

			// Get the distance between p and q
			Point<3,real_number> dr = xa - xb;
			Point<3,real_number> minus_dr = xb - xa;
			// take the norm of this vector
			real_number r2 = norm2(dr);

			// if they interact
			if (r2 < 4.0*H*H)
			{
				real_number r = sqrt(r2);

				Point<3,real_number> dv = va - vb;
				Point<3,real_number> minus_dv = vb - va;

				Point<3,real_number> DW;
				DWab(dr,DW,r,false);

				Point<3,real_number> minus_DW;
				DWab(minus_dr,minus_DW,r,false);

				real_number 	  factor = - massb*((vd.template getProp<Pressure>(a) + vd.template getProp<Pressure>(b)) / (rhoa * rhob) + Tensile(r,rhoa,rhob,Pa,Pb,W_dap) + Pi(dr,r2,dv,rhoa,rhob,massb,cbar,max_visc));
				real_number minus_factor = - massa*((vd.template getProp<Pressure>(a) + vd.template getProp<Pressure>(b)) / (rhoa * rhob) + Tensile(r,rhoa,rhob,Pa,Pb,W_dap) + Pi(minus_dr,r2,minus_dv,rhoa,rhob,massa,cbar,max_visc));

				vd.template getProp<force>(a)[0] += factor * DW.get(0);
				vd.template getProp<force>(a)[1] += factor * DW.get(1);
				vd.template getProp<force>(a)[2] += factor * DW.get(2);

				atomicAdd(&vd.template getProp<force>(b)[0], minus_factor * minus_DW.get(0));
				atomicAdd(&vd.template getProp<force>(b)[1], minus_factor * minus_DW.get(1));
				atomicAdd(&vd.template getProp<force>(b)[2], minus_factor * minus_DW.get(2));

				vd.template getProp<drho>(a) += massb*(dv.get(0)*DW.get(0)+dv.get(1)*DW.get(1)+dv.get(2)*DW.get(2));
				atomicAdd(&vd.template getProp<drho>(b), massa*(minus_dv.get(0)*minus_DW.get(0)+minus_dv.get(1)*minus_DW.get(1)+minus_dv.get(2)*minus_DW.get(2)));
			}

			++Np;
		}

		vd.template getProp<red>(a) = max_visc;
	}
}

template<typename CellList> inline void calc_forces(particles & vd, CellList & NN, real_number & max_visc, size_t cnt)
{
	auto part = vd.getDomainIteratorGPU(32);

	// Update the cell-list
	vd.updateCellListGPU(NN);

	//calc_forces_gpu<<<part.wthr,part.thr>>>(vd.toKernel(),NN.toKernel(),W_dap,cbar);
	CUDA_LAUNCH(calc_forces_gpu,part,vd.toKernel(),NN.toKernel(),W_dap,cbar);

	max_visc = reduce_local<red,_max_>(vd);
}

template<typename vector_type>
__global__ void max_acceleration_and_velocity_gpu(vector_type vd)
{
	auto a = GET_PARTICLE(vd);

	Point<3,real_number> acc(vd.template getProp<force>(a));
	vd.template getProp<red>(a) = norm(acc);

	Point<3,real_number> vel(vd.template getProp<velocity>(a));
	vd.template getProp<red2>(a) = norm(vel);
}

void max_acceleration_and_velocity(particles & vd, real_number & max_acc, real_number & max_vel)
{
	// Calculate the maximum acceleration
	auto part = vd.getDomainIteratorGPU();

	// max_acceleration_and_velocity_gpu<<<part.wthr,part.thr>>>(vd.toKernel());
	CUDA_LAUNCH(max_acceleration_and_velocity_gpu,part,vd.toKernel());

	max_acc = reduce_local<red,_max_>(vd);
	max_vel = reduce_local<red2,_max_>(vd);

	Vcluster<> & v_cl = create_vcluster();
	v_cl.max(max_acc);
	v_cl.max(max_vel);
	v_cl.execute();
}


real_number calc_deltaT(particles & vd, real_number ViscDtMax)
{
	real_number Maxacc = 0.0;
	real_number Maxvel = 0.0;
	max_acceleration_and_velocity(vd,Maxacc,Maxvel);

	//-dt1 depends on force per unit mass.
	const real_number dt_f = (Maxacc)?sqrt(H/Maxacc):std::numeric_limits<int>::max();

	//-dt2 combines the Courant and the viscous time-step controls.
	const real_number dt_cv = H/(std::max(cbar,Maxvel*10.f) + H*ViscDtMax);

	//-dt new value of time step.
	real_number dt=real_number(CFLnumber)*std::min(dt_f,dt_cv);
	if(dt<real_number(DtMin))
	{dt=real_number(DtMin);}

	return dt;
}

template<typename vector_dist_type>
__global__ void verlet_int_gpu(vector_dist_type vd, real_number dt, real_number dt2, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position / Calcula desplazamiento y actualiza posicion.
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];

    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity_prev>(a)[0] + vd.template getProp<force>(a)[0]*dt2;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity_prev>(a)[1] + vd.template getProp<force>(a)[1]*dt2;
	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity_prev>(a)[2] + vd.template getProp<force>(a)[2]*dt2;
	vd.template getProp<rho>(a) = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);

	//! \cond [mark_to_remove_kernel] \endcond

    // Check if the particle go out of range in space and in density, if they do mark them to remove it later
    if (vd.getPos(a)[0] <  0.000263878 || vd.getPos(a)[1] < 0.000263878 || vd.getPos(a)[2] < 0.000263878 ||
        vd.getPos(a)[0] >  0.000263878+1.59947 || vd.getPos(a)[1] > 0.000263878+0.672972 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {vd.template getProp<red>(a) = 1;}
    else
    {vd.template getProp<red>(a) = 0;}

    //! \cond [mark_to_remove_kernel] \endcond

    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

size_t cnt = 0;

void verlet_int(particles & vd, real_number dt)
{
	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;
	real_number dt2 = dt*2.0;

	// verlet_int_gpu<<<part.wthr,part.thr>>>(vd.toKernel(),dt,dt2,dt205);
	CUDA_LAUNCH(verlet_int_gpu,part,vd.toKernel(),dt,dt2,dt205);

	//! \cond [remove_marked_part] \endcond

	// remove the particles marked
	remove_marked<red>(vd);

	//! \cond [remove_marked_part] \endcond

	// increment the iteration counter
	cnt++;
}

template<typename vector_type>
__global__ void euler_int_gpu(vector_type vd,real_number dt, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position / Calcula desplazamiento y actualiza posicion.
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];
    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity>(a)[0] + vd.template getProp<force>(a)[0]*dt;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity>(a)[1] + vd.template getProp<force>(a)[1]*dt;
   	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity>(a)[2] + vd.template getProp<force>(a)[2]*dt;
   	vd.template getProp<rho>(a) = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);

    // Check if the particle go out of range in space and in density
    if (vd.getPos(a)[0] <  0.000263878 || vd.getPos(a)[1] < 0.000263878 || vd.getPos(a)[2] < 0.000263878 ||
        vd.getPos(a)[0] >  0.000263878+1.59947 || vd.getPos(a)[1] > 0.000263878+0.672972 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {vd.template getProp<red>(a) = 1;}
    else
    {vd.template getProp<red>(a) = 0;}

    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

void euler_int(particles & vd, real_number dt)
{

	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;

	// euler_int_gpu<<<part.wthr,part.thr>>>(vd.toKernel(),dt,dt205);
	CUDA_LAUNCH(euler_int_gpu,part,vd.toKernel(),dt,dt205);

	// remove the particles
	remove_marked<red>(vd);

	cnt++;
}

template<typename vector_type, typename NN_type>
__global__ void sensor_pressure_gpu(vector_type vd, NN_type NN, Point<3,real_number> probe, real_number * press_tmp)
{
	real_number tot_ker = 0.0;

	// Get the position of the probe i
	Point<3,real_number> xp = probe;

	// get the iterator over the neighbohood particles of the probes position
	auto itg = NN.getNNIterator(NN.getCell(xp));
	while (itg.isNext())
	{
		auto q = itg.get();

		// Only the fluid particles are importants
		if (vd.template getProp<type>(q) != FLUID)
		{
			++itg;
			continue;
		}

		// Get the position of the neighborhood particle q
		Point<3,real_number> xq = vd.getPos(q);

		// Calculate the contribution of the particle to the pressure
		// of the probe
		real_number r = sqrt(norm2(xp - xq));

		real_number ker = Wab(r) * (MassFluid / rho_zero);

		// Also keep track of the calculation of the summed
		// kernel
		tot_ker += ker;

		// Add the total pressure contribution
		*press_tmp += vd.template getProp<Pressure>(q) * ker;

		// next neighborhood particle
		++itg;
	}

	// We calculate the pressure normalizing the
	// sum over all kernels
	if (tot_ker == 0.0)
	{*press_tmp = 0.0;}
	else
	{*press_tmp = 1.0 / tot_ker * *press_tmp;}
}

template<typename Vector, typename CellList>
inline void sensor_pressure(Vector & vd,
                            CellList & NN,
                            openfpm::vector<openfpm::vector<real_number>> & press_t,
                            openfpm::vector<Point<3,real_number>> & probes)
{
    Vcluster<> & v_cl = create_vcluster();

    press_t.add();

    for (size_t i = 0 ; i < probes.size() ; i++)
    {
    	// A float variable to calculate the pressure of the problem
    	CudaMemory press_tmp_(sizeof(real_number));
    	real_number press_tmp;

        // if the probe is inside the processor domain
		if (vd.getDecomposition().isLocal(probes.get(i)) == true)
		{
			// sensor_pressure_gpu<<<1,1>>>(vd.toKernel(),NN.toKernel(),probes.get(i),(real_number *)press_tmp_.toKernel());
			CUDA_LAUNCH_DIM3(sensor_pressure_gpu,1,1,vd.toKernel(),NN.toKernel(),probes.get(i),(real_number *)press_tmp_.toKernel());

			// move calculated pressure on
			press_tmp_.deviceToHost();
			press_tmp = *(real_number *)press_tmp_.getPointer();
		}

		// This is not necessary in principle, but if you
		// want to make all processor aware of the history of the calculated
		// pressure we have to execute this
		v_cl.sum(press_tmp);
		v_cl.execute();

		// We add the calculated pressure into the history
		press_t.last().add(press_tmp);
	}
}

int main(int argc, char* argv[])
{
    // initialize the library
	openfpm_init(&argc,&argv);

	// It contain for each time-step the value detected by the probes
	openfpm::vector<openfpm::vector<real_number>> press_t;
	openfpm::vector<Point<3,real_number>> probes;

	probes.add({0.8779,0.3,0.02});
	probes.add({0.754,0.31,0.02});

	// Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
	Box<3,real_number> domain({-0.05,-0.05,-0.05},{1.7010,0.7065,0.5025});
	size_t sz[3] = {207,90,66};

	// Fill W_dap
	W_dap = 1.0/Wab(H/1.5);

	// Here we define the boundary conditions of our problem
    size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	// extended boundary around the domain, and the processor domain
	Ghost<3,real_number> g(2*H);

	particles vd(0,domain,bc,g,DEC_GRAN(512));

	//! \cond [draw fluid] \endcond

	// You can ignore all these dp/2.0 is a trick to reach the same initialization
	// of Dual-SPH that use a different criteria to draw particles
	Box<3,real_number> fluid_box({dp/2.0f,dp/2.0f,dp/2.0f},{0.4f+dp/2.0f,0.67f-dp/2.0f,0.3f+dp/2.0f});

	// return an iterator to the fluid particles to add to vd
	auto fluid_it = DrawParticles::DrawBox(vd,sz,domain,fluid_box);

	// here we fill some of the constants needed by the simulation
	max_fluid_height = fluid_it.getBoxMargins().getHigh(2);
	h_swl = fluid_it.getBoxMargins().getHigh(2) - fluid_it.getBoxMargins().getLow(2);
	B = (coeff_sound)*(coeff_sound)*gravity*h_swl*rho_zero / gamma_;
	cbar = coeff_sound * sqrt(gravity * h_swl);

	// for each particle inside the fluid box ...
	while (fluid_it.isNext())
	{
		// ... add a particle ...
		vd.add();

		// ... and set it position ...
		vd.getLastPos()[0] = fluid_it.get().get(0);
		vd.getLastPos()[1] = fluid_it.get().get(1);
		vd.getLastPos()[2] = fluid_it.get().get(2);

		// and its type.
		vd.template getLastProp<type>() = FLUID;

		// We also initialize the density of the particle and the hydro-static pressure given by
		//
		// rho_zero*g*h = P
		//
		// rho_p = (P/B + 1)^(1/Gamma) * rho_zero
		//

		vd.template getLastProp<Pressure>() = rho_zero * gravity *  (max_fluid_height - fluid_it.get().get(2));

		vd.template getLastProp<rho>() = pow(vd.template getLastProp<Pressure>() / B + 1, 1.0/gamma_) * rho_zero;
		vd.template getLastProp<rho_prev>() = vd.template getLastProp<rho>();
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		// next fluid particle
		++fluid_it;
	}

	// Recipient
	Box<3,real_number> recipient1({0.0f,0.0f,0.0f},{1.6f+dp/2.0f,0.67f+dp/2.0f,0.4f+dp/2.0f});
	Box<3,real_number> recipient2({dp,dp,dp},{1.6f-dp/2.0f,0.67f-dp/2.0f,0.4f+dp/2.0f});

	Box<3,real_number> obstacle1({0.9f,0.24f-dp/2.0f,0.0f},{1.02f+dp/2.0f,0.36f,0.45f+dp/2.0f});
	Box<3,real_number> obstacle2({0.9f+dp,0.24f+dp/2.0f,0.0f},{1.02f-dp/2.0f,0.36f-dp,0.45f-dp/2.0f});
	Box<3,real_number> obstacle3({0.9f+dp,0.24f,0.0f},{1.02f,0.36f,0.45f});

	openfpm::vector<Box<3,real_number>> holes;
	holes.add(recipient2);
	holes.add(obstacle1);
	auto bound_box = DrawParticles::DrawSkin(vd,sz,domain,holes,recipient1);

	while (bound_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = bound_box.get().get(0);
		vd.getLastPos()[1] = bound_box.get().get(1);
		vd.getLastPos()[2] = bound_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++bound_box;
	}

	auto obstacle_box = DrawParticles::DrawSkin(vd,sz,domain,obstacle2,obstacle1);

	while (obstacle_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = obstacle_box.get().get(0);
		vd.getLastPos()[1] = obstacle_box.get().get(1);
		vd.getLastPos()[2] = obstacle_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++obstacle_box;
	}
	
	vd.map();

	// Now that we fill the vector with particles
	ModelCustom md;

	vd.addComputationCosts(md);
	vd.getDecomposition().decompose();
	vd.map();
    
	///////////////////////////

	// Ok the initialization is done on CPU on GPU we are doing the main loop, so first we offload all properties on GPU

	vd.hostToDevicePos();
	vd.template hostToDeviceProp<type,rho,rho_prev,Pressure,velocity,velocity_prev>();

	vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);

	auto NN = vd.getCellListGPU(2*H, 2);

	timer tot_sim;
	tot_sim.start();

	size_t write = 0;
	size_t it = 0;
	size_t it_reb = 0;
	real_number t = 0.0;
	size_t timestep = 0;

	while (timestep < 1000)
	{
		Vcluster<> & v_cl = create_vcluster();
		timer it_time;

		////// Do rebalancing every 200 timesteps
		it_reb++;
		if (it_reb == 300)
		{
			vd.map(RUN_ON_DEVICE);

			it_reb = 0;
			ModelCustom md;
			vd.addComputationCosts(md);
			vd.getDecomposition().decompose();

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "REBALANCED " << it_reb << std::endl;}
		}

		vd.map(RUN_ON_DEVICE);

		// Calculate pressure from the density
		EqState(vd);

		real_number max_visc = 0.0;

		vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);

		// Calc forces
		calc_forces(vd,NN,max_visc,cnt);

		// Get the maximum viscosity term across processors
		v_cl.max(max_visc);
		v_cl.execute();

		// Calculate delta t integration
		real_number dt = calc_deltaT(vd,max_visc);

		// VerletStep or euler step
		it++;
		if (it < 40)
			verlet_int(vd,dt);
		else
		{
			euler_int(vd,dt);
			it = 0;
		}

		t += dt;

		if (timestep % 100 == 0)
		{
			// Sensor pressure require update ghost, so we ensure that particles are distributed correctly
			// and ghost are updated
			vd.map(RUN_ON_DEVICE);
			vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);
			vd.updateCellListGPU(NN);

			// calculate the pressure at the sensor points
			//sensor_pressure(vd,NN,press_t,probes);

			std::cout << "OUTPUT " << dt << std::endl;

			// When we write we have move all the particles information back to CPU

			vd.deviceToHostPos();
			vd.deviceToHostProp<type,rho,rho_prev,Pressure,drho,force,velocity,velocity_prev,red,red2>();

			vd.write_frame("Geometry",write);
			write++;

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  write " << it_time.getwct() << "   " << it_reb << "   " << cnt << " Max visc: " << max_visc << "   " << vd.size_local()  << std::endl;}
		}
		//else
		//{
		//	if (v_cl.getProcessUnitID() == 0)
		//	{std::cout << "TIME: " << t << "  " << it_time.getwct() << "   " << it_reb << "   " << cnt  << " Max visc: " << max_visc << "   " << vd.size_local() << std::endl;}
		//}
		++timestep;
	}

	tot_sim.stop();
	std::cout << "Time to complete: " << tot_sim.getwct() << " seconds" << std::endl;

	openfpm_finalize();
}
 
#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif
